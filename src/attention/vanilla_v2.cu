#include "hip/hip_runtime.h"
#include "attention.hpp"
#include "cuda_utils.hpp"

#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_math_constants.h>

namespace cg = cooperative_groups;

#define PARTIALS_PTR(arr, bh_idx, row, seq_len, blocks_x)                                          \
    ((arr) + ROW_OFFSET(bh_idx, row, seq_len, blocks_x))

#define TILE_DIM 32

// Kernel: strided and batched QK^T with partial max reductions only
__global__ void qk_dot_partial_reduce_v2(const float *__restrict__ Q, const float *__restrict__ K,
                                         float *attn_scores, float *row_max_partials, int seq_len,
                                         int head_dim, const float scale) {

    // cooperative-groups block object
    cg::thread_block cg_block = cg::this_thread_block();
    // create a warp-sized tile group
    auto warp = cg::tiled_partition<TILE_DIM>(cg_block);

    int bh_idx = blockIdx.z;
    int blocks_x = gridDim.x;
    int block_row = blockIdx.y * TILE_DIM;
    int block_col = blockIdx.x * TILE_DIM;

    int local_row = threadIdx.y;
    int local_col = threadIdx.x;

    int row = block_row + local_row;
    int col = block_col + local_col;

    const float *Qbh_base = Q_PTR(Q, bh_idx, seq_len, head_dim);
    const float *Kbh_base = K_PTR(K, bh_idx, seq_len, head_dim);
    float *attn_bh_base = ATTN_PTR(attn_scores, bh_idx, seq_len);

    __shared__ float Q_tile[TILE_DIM][TILE_DIM];
    __shared__ float Kt_tile[TILE_DIM][TILE_DIM + 1]; // padding

    float acc = 0.0f;

    // per-thread workload
    int num_tiles = (head_dim + TILE_DIM - 1) / TILE_DIM;
    for (int t = 0; t < num_tiles; ++t) {
        int q_col = t * TILE_DIM + local_col;
        int k_row = t * TILE_DIM + local_row;

        float q_val = (row < seq_len && q_col < head_dim) ? Qbh_base[row * head_dim + q_col] : 0.0f;
        Q_tile[local_row][local_col] = q_val;

        // note transpose
        float kt_val =
            (col < seq_len && k_row < head_dim) ? Kbh_base[col * head_dim + k_row] : 0.0f;
        Kt_tile[local_col][local_row] = kt_val;

        cg_block.sync();

        // inner product for this tile vectorized in groups of 4
#pragma unroll
        for (int k = 0; k < TILE_DIM; k += 4) {
            float4 q_vec;
            q_vec.x = Q_tile[local_row][k + 0];
            q_vec.y = Q_tile[local_row][k + 1];
            q_vec.z = Q_tile[local_row][k + 2];
            q_vec.w = Q_tile[local_row][k + 3];

            float4 k_vec;
            k_vec.x = Kt_tile[local_col][k + 0];
            k_vec.y = Kt_tile[local_col][k + 1];
            k_vec.z = Kt_tile[local_col][k + 2];
            k_vec.w = Kt_tile[local_col][k + 3];

            acc = fmaf(q_vec.x, k_vec.x, acc);
            acc = fmaf(q_vec.y, k_vec.y, acc);
            acc = fmaf(q_vec.z, k_vec.z, acc);
            acc = fmaf(q_vec.w, k_vec.w, acc);
        }
        cg_block.sync();
    }

    acc *= scale;

    if (row < seq_len && col < seq_len) {
        attn_bh_base[row * seq_len + col] = acc;
    }

    int warps_per_row = (blockDim.x + warp.size() - 1) / warp.size();

    // per-warp max scores i.e each thread in a warp will hold the same max value
    float max_score = acc;
    for (int offset = warp.size() / 2; offset > 0; offset >>= 1) {
        float other = warp.shfl_xor(max_score, offset);
        max_score = fmaxf(max_score, other);
    }

    if (warps_per_row > 1) {
        extern __shared__ float warp_max_scores[];
        int warp_id_row = threadIdx.y;
        int warp_id_col = threadIdx.x / warp.size();
        // thread 0 per-warp loads its max score
        if (warp.thread_rank() == 0) {
            warp_max_scores[warp_id_row * warps_per_row + warp_id_col] = max_score;
        }
        cg_block.sync();

        // block-level reduction for max scores per row of a block
        if (warp_id_col == 0) {
            int warp_col_fetch_id = warp.thread_rank();
            float val = (warp_col_fetch_id < warps_per_row)
                            ? warp_max_scores[warp_id_row * warps_per_row + warp_col_fetch_id]
                            : -HIP_INF_F;
            for (int offset = warp.size() / 2; offset > 0; offset >>= 1) {
                float other = warp.shfl_xor(val, offset);
                max_score = fmaxf(val, other);
            }
            max_score = val;
        }
        cg_block.sync();
    }

    // write per-block row partial maxes to global memory from the first column in tile (local_col
    // == 0)
    if (local_col == 0 && row < seq_len) {
        int idx = ROW_OFFSET(bh_idx, row, seq_len, blocks_x) + blockIdx.x;
        row_max_partials[idx] = max_score;
    }
}

// Kernel: find global max and compute global sum
__global__ void softmax_inplace_v2(float *attention_scores,
                                   const float *__restrict__ row_max_partials, int seq_len,
                                   int partials_blocks_x) {

    int bh_idx = blockIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.x;

    if (row >= seq_len)
        return;

    const float *row_max_bh_base =
        PARTIALS_PTR(row_max_partials, bh_idx, row, seq_len, partials_blocks_x);

    // Find max (for numerical stability)
    float row_max;
    __shared__ float shared_row_max;
    if (tid == 0) {
        row_max = -HIP_INF_F;
        for (int bx = 0; bx < partials_blocks_x; bx++) {
            float block_max = row_max_bh_base[bx];
            if (block_max > row_max)
                row_max = block_max;
        }
        shared_row_max = row_max;
    }
    // synchronize row max from thread 0 to all
    __syncthreads();
    row_max = shared_row_max;

    // Compute sum of exp(score - row_max) across all columns
    float *attn_bh_base = ATTN_PTR(attention_scores, bh_idx, seq_len);
    float row_sum = 0.0f;

    for (int col = tid; col < seq_len; col += blockDim.x) {
        int idx = row * seq_len + col;
        float score = expf(attn_bh_base[idx] - row_max);
        row_sum += score;
        attn_bh_base[idx] = score;
    }

    extern __shared__ float shared_sum[];
    shared_sum[tid] = row_sum;
    __syncthreads();
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tid < offset)
            shared_sum[tid] += shared_sum[tid + offset];
        __syncthreads();
    }

    // normalize attention scores
    const float norm = 1 / shared_sum[0];
    for (int col = tid; col < seq_len; col += blockDim.x) {
        attn_bh_base[row * seq_len + col] *= norm;
    }
}

// Kernel: apply softmax and multiply by V
__global__ void softmax_multV_v2(const float *__restrict__ attention_scores,
                                 const float *__restrict__ V, float *O, int seq_len, int head_dim) {

    int bh_idx = blockIdx.z;
    int block_row = blockIdx.y * TILE_DIM;
    int block_col = blockIdx.x * TILE_DIM;

    int local_row = threadIdx.y;
    int local_col = threadIdx.x;

    int row = block_row + local_row;
    int col = block_col + local_col;

    if (row >= seq_len)
        return;

    const float *attn_bh_base = ATTN_PTR(attention_scores, bh_idx, seq_len);
    const float *Vbh_base = V_PTR(V, bh_idx, seq_len, head_dim);
    float *Obh_base = O_PTR(O, bh_idx, seq_len, head_dim);

    __shared__ float softmax_tile[TILE_DIM][TILE_DIM];
    __shared__ float V_tile[TILE_DIM][TILE_DIM];

    float acc = 0.0f;

    int num_tiles = (seq_len + TILE_DIM - 1) / TILE_DIM;

    for (int t = 0; t < num_tiles; ++t) {
        int k = t * TILE_DIM + local_col;

        // Load softmax tile: softmax[row, k]
        if (row < seq_len && k < seq_len) {
            softmax_tile[local_row][local_col] = attn_bh_base[row * seq_len + k];
        } else {
            softmax_tile[local_row][local_col] = 0.0f;
        }

        int k_v = t * TILE_DIM + local_row;
        if (k_v < seq_len && col < head_dim) {
            V_tile[local_row][local_col] = Vbh_base[k_v * head_dim + col];
        } else {
            V_tile[local_row][local_col] = 0.0f;
        }
        __syncthreads();

#pragma unroll
        for (int k = 0; k < TILE_DIM; ++k) {
            acc += softmax_tile[local_row][k] * V_tile[k][local_col];
        }
        __syncthreads();
    }

    if (row < seq_len && col < head_dim) {
        Obh_base[row * head_dim + col] = acc;
    }
}

// Simple attention implementation
struct VanillaAttentionV2 : public Attention {

    void forward(const float *Q, const float *K, const float *V, float *O, uint32_t batch_size,
                 uint32_t num_heads, uint32_t seq_len, uint32_t head_dim) override {

        float *attention_scores;
        float *row_max_partials;

        const float scale = 1.0f / sqrtf((float) head_dim);
        const size_t n_qkt = batch_size * num_heads * seq_len * seq_len;
        hipMalloc(&attention_scores, n_qkt * sizeof(float));

        int blocks_x = (seq_len + TILE_DIM - 1) / TILE_DIM;
        int blocks_y = (seq_len + TILE_DIM - 1) / TILE_DIM;

        const size_t n_partials = batch_size * num_heads * seq_len * blocks_x;
        hipMalloc(&row_max_partials, n_partials * sizeof(float));

        // Kernel 1: Compute QK^T and partial max values
        dim3 threads(TILE_DIM, TILE_DIM);
        dim3 grid(blocks_x, blocks_y, batch_size * num_heads);
        const int num_warps_in_x = TILE_DIM / 32;
        int shared_bytes = num_warps_in_x * sizeof(float);
        qk_dot_partial_reduce_v2<<<grid, threads, shared_bytes>>>(
            Q, K, attention_scores, row_max_partials, seq_len, head_dim, scale);
        hipDeviceSynchronize();
        CUDA_CHECK();

        // save_device_ptr_as_buffer("QKt.bin", attention_scores, n_qkt);
        // Kernel 2:  Apply per row max and normalize with per row sum
        int device_max_threads;
        hipDeviceGetAttribute(&device_max_threads, hipDeviceAttributeMaxThreadsPerBlock, 0);
        int per_row_threads = std::min(TILE_DIM * blocks_x, device_max_threads);
        per_row_threads = std::max(32, per_row_threads);
        dim3 threads2(per_row_threads, 1, 1);
        dim3 grid2(1, seq_len, batch_size * num_heads);
        shared_bytes = per_row_threads * sizeof(float);
        softmax_inplace_v2<<<grid2, threads2, shared_bytes>>>(attention_scores, row_max_partials,
                                                              seq_len, blocks_x);
        hipDeviceSynchronize();
        CUDA_CHECK();

        // Kernel 3: Apply softmax and multiply by V
        softmax_multV_v2<<<grid, threads>>>(attention_scores, V, O, seq_len, head_dim);
        hipDeviceSynchronize();
        CUDA_CHECK();

        hipFree(attention_scores);
        hipFree(row_max_partials);
        CUDA_CHECK();
    }
};

// Factory function
extern "C" Attention *create_vanilla_attention_v2() {
    return new VanillaAttentionV2();
}